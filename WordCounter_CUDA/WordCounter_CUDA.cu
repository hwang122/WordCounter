#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "simpleHashTable.h"

#define BUFFER_SIZE		SIZE * 6

//global function callable from main function
//count each word's occurrence
__global__ void counter(unsigned int *key, Table table, Lock *lock, int count)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
	
	/** initialize table's pool in the initialize table funtion
		make sure zero out values could only occurs once
	//initialize table
	if(tid == 0)
	{
		printf("In zero out table\n");
		zero_out_values_in_table(table);
		//need to be synchronized
		__syncthreads();
	}
	**/
	//all the key with index larger than count is not exist
    while (tid < count) {
		unsigned int k = key[tid];
		put(table, k, lock, tid);
        tid += stride;
    }
}

int main(int argc, char* argv[])
{
	if(argc != 4)
	{
		printf("Usage: %s <File name> <Searching word> <Number of threads>",
				argv[0]);
		exit(-1);
	}
	char *filename = argv[1];
	unsigned int key_num = (unsigned int)atoi(argv[2]);
	int numThread = atoi(argv[3]);
	
	//open file
	FILE *pFile;
	pFile = fopen(filename, "r");
	if(pFile == NULL)
	{
		printf("Fail to open file");
		exit(-1);
	}
	//time function
	hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));
	
	//copy hash table and lock to GPU
	Table dev_table;
    initialize_table(dev_table, HASH_ENTRIES, ELEMENTS);

    Lock lock[HASH_ENTRIES];
    Lock *dev_lock;
    HANDLE_ERROR(hipMalloc((void**)&dev_lock, HASH_ENTRIES * sizeof( Lock)));
    HANDLE_ERROR(hipMemcpy(dev_lock, lock, HASH_ENTRIES * sizeof( Lock ),	
                              hipMemcpyHostToDevice ) );
	
	/**
	//set count in GPU to count number of elements
	unsigned int count = 0, *dev_count;
	hipMalloc((void**)&dev_count, sizeof(unsigned int));
	HANDLE_ERROR(hipMemcpy(dev_count, &count, sizeof(unsigned int), 
							hipMemcpyHostToDevice));
	**/
	
	//set block number and thread number per block
	int numBlock = 32;
	int threadPerBlock = numThread/numBlock;
	//copy file to GPU
	unsigned int *key, *d_key;
	while(!feof(pFile))
	{
		key = (unsigned int*)malloc(BUFFER_SIZE);
		int num = BUFFER_SIZE/sizeof(unsigned int);
		
		int i, count;
		for(i = 0; i < num && !feof(pFile); i++)
		{
			fscanf(pFile, "%u", &key[i]);
			count = i;
		}
		
		HANDLE_ERROR(hipMalloc((void**)&d_key, BUFFER_SIZE));
		HANDLE_ERROR(hipMemcpy(d_key, key, BUFFER_SIZE, hipMemcpyHostToDevice));

		//count the word on GPU
		counter<<<numBlock, threadPerBlock>>>(d_key, dev_table, dev_lock, count+1);
		//wait until all device ended
		hipDeviceSynchronize();
		//free memory in GPU
		HANDLE_ERROR(hipFree(d_key));
		free(key);
	}
	
	//copy hash table from device to host
	Table table;
	copy_table_to_host(dev_table, table);
	unsigned long occurrence = 0;
	occurrence = get(table, key_num);
	if(occurrence == 0)
		printf("Key number not found!");
	else
		printf("Occurrence of Key number: %u\n", occurrence);
	
	HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Time to hash:  %3.1f ms\n", elapsedTime);
	
	free(table.pool);
	free(table.entries);
	HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    free_table(dev_table);
    HANDLE_ERROR(hipFree(dev_lock));
	return 0;
}

